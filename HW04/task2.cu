#include "stencil.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <random>
#include <vector>
#include <chrono>

using namespace std;
int main(int argc, char *argv[]){

    // Get Command Line Input
    int n = atoi(argv[1]);
    int R = atoi(argv[2]);
    int threads_per_block = atoi(argv[3]);

    // Initialize Arrays on the Host
    float* image = new float[n];
    float* output = new float[n];
    float* mask = new float[2*R+1];

    // Initialize Arrays on the Device
    float *imaged = NULL;
    float *outputd = NULL;
    float *maskd = NULL;
    hipMalloc(&imaged, n*sizeof(float));
    hipMalloc(&outputd, n*sizeof(float));
    hipMalloc(&maskd, (2*R+1)*sizeof(float));

    // Initialization for randomization
    random_device entropy_source;
    mt19937 generator(entropy_source());
    
    // Generate random float values and populate host arrays
    uniform_real_distribution<float> RD(-1.0,1.0);
    for (int i = 0; i<n; i++){
        image[i] = RD(generator);
    }
    for (int j = 0; j<(2*R+1);j++){
        mask[j] = RD(generator);
    }
    // Initialization for CUDA Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copy Randomized Arrays from Host to Device
    hipMemcpy(imaged,image,n*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(maskd,mask,(2*R+1)*sizeof(float),hipMemcpyHostToDevice);


    // Call and time stencil function
    hipEventRecord(start);
    stencil(imaged,maskd,outputd,n,R,threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate time taken
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    // Copy Results from device to host memory
    hipMemcpy(output,outputd,n*sizeof(float),hipMemcpyDeviceToHost);

    // Print Results
    cout << output[n-1] << "\n";
    cout << ms << "\n";

    // Deallocate memory
    hipFree(outputd);
    hipFree(imaged);
    hipFree(maskd);
    delete[] output;
    delete[] image;
    delete[] mask;
}
