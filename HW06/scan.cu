#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <iostream>
#include <cstddef>


// Performs an *inclusive scan* on the array input and writes the results to the array output.
// The scan should be computed by making calls to your kernel hillis_steele with
// threads_per_block threads per block in a 1D configuration.
// input and output are arrays of length n allocated as managed memory.
//
// Assumptions:
// - n <= threads_per_block * threads_per_block

__global__ void hillis_steele(const float *input, float *output, float *sumarray, int n){
    extern volatile __shared__ float sharedarray[];
    // get thread information
    int thread = threadIdx.x; 
    int block = blockIdx.x;
    int blocksize = blockDim.x;
    int pout = 0, pin = 1;
    // Copy input to sharred array
    sharedarray[thread] = input[block*blocksize + thread];
    __syncthreads();
    // Begin iterating through offsets across dual buffers
    for (int offset = 1; offset<blocksize; offset *=2){
        pout = 1 - pout;
        pin = 1 - pout;
        if (thread >= offset) sharedarray[pout*blocksize+thread]= sharedarray[pin*blocksize+thread] + sharedarray[pin*blocksize + thread - offset];
        else sharedarray[pout*blocksize+thread] = sharedarray[pin*blocksize+thread];
        __syncthreads();
    }
    // write scan results of block to output array
    output[block*blocksize + thread] = sharedarray[pout*blocksize+thread];
    // write total sum of block to the blocksum array
    if (thread == blocksize - 1) sumarray[block] = output[block*blocksize + thread];
}
__global__ void hillis_steele_odd(const float *input, float *output, float* sumarray,int n){
    extern volatile __shared__ float sharedarray[];
    // get thread information
    int thread = threadIdx.x;
    int blocksize = blockDim.x;
    int needed_threads = n % blocksize;
    int pout = 0, pin = 1;
    // Copy input to sharred array
    if (thread >= needed_threads){
        sharedarray[thread] = 0;
        __syncthreads();
    }
    else{
        sharedarray[thread] = input[n-needed_threads + thread];
        __syncthreads();
        for (int offset = 1; offset<n; offset *=2){
            pout = 1 - pout;
            pin = 1 - pout;
            if (thread >= offset) sharedarray[pout*blocksize+thread]= sharedarray[pin*blocksize+thread] + sharedarray[pin*blocksize + thread - offset];
            else sharedarray[pout*blocksize+thread] = sharedarray[pin*blocksize+thread];
            __syncthreads();
        }
         // write scanned block to output array
        output[n-needed_threads + thread] = sharedarray[pout*blocksize+thread];
        if (thread == needed_threads - 1) sumarray[n/blocksize] = output[n-needed_threads + thread];
    }
}
__global__ void kernel_add(const float * scannedsumarray, float * output, int n){
    // get thread information
    int thread = threadIdx.x;
    int block = blockIdx.x;
    int blocksize = blockDim.x;
    if (block > 0 && block*blocksize + thread < n){
        output[block*blocksize + thread] += scannedsumarray[block - 1];
    }
}

__host__ void scan(const float* input, float* output, unsigned int n, unsigned int threads_per_block){
    float *sumarray, *scannedsumarray;
    int blocksneeded_main = n/threads_per_block;

// determine size of sumscan array and f helper kernel needed for first scan
if (n % threads_per_block > 0){
    hipMallocManaged((void**)&sumarray, (blocksneeded_main + 1)*sizeof(float));
    hipMallocManaged((void**)&scannedsumarray, (blocksneeded_main + 1)*sizeof(float));
    hillis_steele<<<blocksneeded_main,threads_per_block,2*threads_per_block*sizeof(float)>>>(input,output,sumarray, n);
    hillis_steele_odd<<<1,threads_per_block,2*threads_per_block*sizeof(float)>>>(input,output,sumarray,n);
}
else {
    hipMallocManaged((void**)&sumarray, (blocksneeded_main)*sizeof(float));
    hipMallocManaged((void**)&scannedsumarray, (blocksneeded_main)*sizeof(float));
    hillis_steele<<<blocksneeded_main,threads_per_block,2*threads_per_block*sizeof(float)>>>(input,output,sumarray, n);
}

// determine if the sumarray needs a full block kernel or helper kernel
if (blocksneeded_main == threads_per_block){
    hillis_steele<<<1,threads_per_block,2*threads_per_block*sizeof(float)>>>(sumarray,scannedsumarray,sumarray,blocksneeded);
}
else {
    hillis_steele_odd<<<1,threads_per_block,2*threads_per_block*sizeof(float)>>>(sumarray,scannedsumarray,sumarray,blocksneeded);
}

// Add scanned sum array values to output array
kernel_add<<<blocksneeded_main,threads_per_block>>>(scannedsumarray,output,n);
 
// memory cleanup
hipFree(sumarray);
hipFree(scannedsumarray);
}