#include "mmul.h"
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <random>
#include <vector>
#include <chrono>


int main(int argc, char *argv[]){

    // Get Command Line Input
    int n = atoi(argv[1]);
    unsigned int test_n = atoi(argv[2]);

    // Initialize Managed Arrays
    float *A, *B, *C;
    hipMallocManaged((void**)&A, n*n*sizeof(float));
    hipMallocManaged((void**)&B, n*n*sizeof(float));
    hipMallocManaged((void**)&C, n*n*sizeof(float));

    // Initialization for CUDA Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float t_total, ms, t_avg;

    // Initialization for randomization
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());

    // Generate random float values and populate arrays in Column Major Order
    std::uniform_real_distribution<float> RD(-1.0,1.0);
    for (int i = 0; i<n; i++){
        for (int j = 0; j<n; j++){
            A[j*n+i] = RD(generator);
            B[j*n+i] = RD(generator);
            C[j*n+1] = RD(generator);
        }
    }

    // Intitialization for cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Call and time mmul function test_n times, capturing the time for each call
    for (int i = 0; i < test_n; i++){
        hipEventRecord(start);
        mmul(handle,A,B,C,n);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        // Get function elapsed time
        hipEventElapsedTime(&ms, start, stop);
        t_total += ms;
    }
    // Calculate Total time taken
    t_avg = t_total/test_n;
    
    // Print Results
    prtinf("%f \n",t_avg);

    // Memory Cleanup
    hipblasDestroy(handle);
    hipFree(A);
    hipFree(B);
    hipFree(C);
}