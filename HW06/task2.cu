#include "scan.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <math.h>
#include <random>
#include <vector>
#include <chrono>


int main(int argc, char *argv[]){

    // Get Command Line Input
    int n = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    // Allocate managed memory
    float* input, output; 
    hipMallocManaged((void**)&input, n*sizeof(float));
    hipMallocManaged((void**)&output, n*sizeof(float));

    // Initialization for CUDA Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Initialization for randomization
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    
    // Generate random float values and populate host arrays
    std::uniform_real_distribution<float> RD(-1.0,1.0);
    for (int i = 0; i<n; i++){
        input[i] = RD(generator);
    }

    // Call and time scan function
    hipEventRecord(start);
    scan(input, output, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Get function elapsed time
    float ms;
    hipEventElapsedTime(&ms, start, stop);

