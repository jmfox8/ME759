#include "hip/hip_runtime.h"
#include "single_definitions.cuh"
#include "RK4.cuh"
#include <cstddef>
#include <math.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>

#define PI 3.14159


// CONFIGURED FOR SINGLE INVERTED PENDULUM
int main(int argc, char *argv[]){
    float q0[2], tmax_amp, tmin_amp, tmax_dur, tmin_dur;
    segment vals[2];
    // Get Command Line Input
    //q0[0]= atof(argv[1]);
    //q0[1] = atof(argv[2]);

    // Initialize Variables for timing
    std::chrono::duration<double, std::milli> ms;
    std::chrono::high_resolution_clock::time_point start;
    std::chrono::high_resolution_clock::time_point end;

    q0[0] = -5*PI/180;
    q0[1] = 0*PI/180;

    float h = 0.01; // Step size for path solver [s]
    int t_n = 1000; // Number of values attempted for each torque parameter
    hipMallocManaged((void**)&torque_array, t_n*t_n*sizeof(tpulseinfo));
    hipMallocManaged((void**)&output_bests, t_n*t_n*sizeof(RK4out));
    RK4out overall_best;
    overall_best.norm = 100;
    
    vals[0].l = 0.867; // anthro table length of ankle to hip
    vals[0].lc = 0.589; // anthro table lenth of ankle to CM of legs
    vals[0].m = 26.30; // anthro table mass of lower leg segments
    vals[0].I = 1.4; // anthro table moment of intertia of leg segments
    vals[0].Icm = vals[0].I+vals[0].m*vals[0].lc*vals[0].lc;
    
    if (q0[0] <= 0 )
    {
        tmin_amp = 0;
        tmax_amp = 100;
    }
    else
    {
        tmax_amp = -50;
        tmin_amp = -100;
    }

    tmax_dur = 0.3;
    tmin_dur = 0.05;
    float t_dur_step = (tmax_dur - tmin_dur)/t_n;
    float t_amp_step = (tmax_amp - tmin_amp)/t_n;

    float sim_time = 0.5;
    torque_array[0].amp = tmin_amp;
    torque_array[0].duration = tmin_dur;

    for (int i = 0; i < t_n; i++){
        for (int j = 0; j<t_n; j++){
            torque_array[i*t_n + j].amp = tmin_amp + i*t_amp_step;
            torque_array[i*t_n + j].duration = tmin_dur + j*t_dur_step;
        }
    }
    start = std::chrono::high_resolution_clock::now();
    
        RK4<<<1,256>>(sim_time,h,torque_array[i], q0, vals,output_bests);
        for (int i = 0; i < t_n*t_n; i++){
        if (overall_best.norm > output_bests[i].norm) overall_best = output_bests[i];
    }

    end = std::chrono::high_resolution_clock::now();
    ms = std::chrono::duration_cast<std::chrono::duration<double, std::milli> >(end - start);
    std::cout << "time for loop: "<< ms.count() <<"\n";
    std::cout << "best performance - norm: " << overall_best.norm << "torque amp: " << overall_best.torque.amp << "torque time: " << overall_best.torque.duration << "\n";
}
